#include "THCBlas.h"
#include "THCGeneral.h"

void THCudaBlas_swap(THCState *state, int64 n, float *x, int64 incx, float *y, int64 incy)
{
  if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    THCublasCheck(hipblasSswap(THCState_getCurrentBlasHandle(state), i_n, x, i_incx, y, i_incy));
    return;
  }
  THError("Cublas_swap only supports n, incx and"
          " incy upto signed integer limits: %d", INT_MAX);
}

void THCudaBlas_scal(THCState *state, int64 n, float a, float *x, int64 incx)
{
  if(n == 1)
    incx = 1;

  if( (n <= INT_MAX) && (incx <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    THCublasCheck(hipblasSscal(THCState_getCurrentBlasHandle(state), i_n, &a, x, i_incx));
    return;
  }
  THError("Cublas_scal only supports n and incx "
          "upto signed integer limits: %d", INT_MAX);
}

void THCudaBlas_copy(THCState *state, int64 n, float *x, int64 incx, float *y, int64 incy)
{
  if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    THCublasCheck(hipblasScopy(THCState_getCurrentBlasHandle(state), i_n, x, i_incx, y, i_incy));
    return;
  }

  THError("Cublas_copy only supports n, incx and incy "
          "upto signed integer limits: %d", INT_MAX);
}

void THCudaBlas_axpy(THCState *state, int64 n, float a, float *x, int64 incx, float *y, int64 incy)
{
    if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    THCublasCheck(hipblasSaxpy(THCState_getCurrentBlasHandle(state), i_n, &a, x, i_incx, y, i_incy));
    return;
  }

  THError("Cublas_axpy only supports n, incx and incy "
          "upto signed integer limits: %d", INT_MAX);
}

float THCudaBlas_dot(THCState *state, int64 n, float *x, int64 incx, float *y, int64 incy)
{
  if(n == 1)
  {
    incx = 1;
    incy = 1;
  }

  if( (n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX) )
  {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    THCublasCheck(hipblasSdot(THCState_getCurrentBlasHandle(state), i_n, x, i_incx, y, i_incy, &result));
    hipDeviceSynchronize();
    return result;
  }
  THError("Cublas_dot only supports n, incx and incy "
          "upto signed integer limits: %d", INT_MAX);
  return -1;
}

/* Level 2 */
void THCudaBlas_gemv(THCState *state, char trans, int64 m, int64 n, float alpha, float *a, int64 lda, float *x, int64 incx, float beta, float *y, int64 incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    THCublasCheck(hipblasSgemv(THCState_getCurrentBlasHandle(state), op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_gemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_ger(THCState *state, int64 m, int64 n, float alpha, float *x, int64 incx, float *y, int64 incy, float *a, int64 lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      THCublasCheck(hipblasSger(THCState_getCurrentBlasHandle(state), i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_ger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

hipblasOperation_t convertTransToCublasOperation(char trans) {
  if (trans == 't') return HIPBLAS_OP_T;
  else if (trans == 'n') return HIPBLAS_OP_N;
  else if (trans == 'c') return HIPBLAS_OP_C;
  else {
    THError("trans must be one of: t, n, c");
    return HIPBLAS_OP_T;
  }
}

void adjustLd(char transa, char transb, int64 m, int64 n, int64 k, int64 *lda, int64 *ldb, int64 *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  if(n == 1)
    *ldc = m;

  if(transa_)
  {
    if(m == 1)
      *lda = k;
  }
  else
  {
    if(k == 1)
      *lda = m;
  }

  if(transb_)
  {
    if(k == 1)
      *ldb = n;
  }
  else
  {
    if(n == 1)
      *ldb = k;
  }
}

/* Level 3 */
void THCudaBlas_gemm(THCState *state, char transa, char transb, int64 m, int64 n, int64 k, float alpha, float *a, int64 lda, float *b, int64 ldb, float beta, float *c, int64 ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    THCublasCheck(hipblasSgemm(THCState_getCurrentBlasHandle(state), opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_gemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_gemmBatched(THCState *state, char transa, char transb, int64 m, int64 n, int64 k,
                            float alpha, const float *a[], int64 lda, const float *b[], int64 ldb,
                            float beta, float *c[], int64 ldc, int64 batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_gemm only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  THCublasCheck(hipblasSgemmBatched(THCState_getCurrentBlasHandle(state),
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}
