#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

struct TensorAddConstantOp {
  TensorAddConstantOp(float v) : val(v) {}
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = *in + val;
  }

  __device__ __forceinline__ void operator()(float* v) {
    *v += val;
  }

  const float val;
};

void THCudaTensor_add(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorAddConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src_);

    if (!THCudaTensor_pointwiseApply2(state, self_, src_, TensorAddConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorMulConstantOp {
  TensorMulConstantOp(float v) : val(v) {}
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = *in * val;
  }

  __device__ __forceinline__ void operator()(float* v) {
    *v *= val;
  }

  const float val;
};

void THCudaTensor_mul(THCState *state, THCudaTensor *self_, THCudaTensor *src_, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src_);

    if (!THCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_div(THCState* state, THCudaTensor *self_, THCudaTensor *src_, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(value != 0.0f, 3, "divide by zero");

  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorMulConstantOp(1.0f / value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src_);

    if (!THCudaTensor_pointwiseApply2(state, self_, src_, TensorMulConstantOp(1.0f / value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

template <int Upper>
struct TensorTriOp {
  TensorTriOp(float *start_, int64 stride0_, int64 stride1_, int64 k_)
    : start(start_), stride0(stride0_), stride1(stride1_), k(k_) {}

  __device__ __forceinline__ int mask(float *in) {
    ptrdiff_t n = in - start;
    int64 row, col;
    if (stride0 > stride1)
    {
      row = (int64) (n / stride0);
      col = (int64) ((n % stride0) / stride1);
    }
    else
    {
      row = (int64) ((n % stride1) / stride0);
      col = (int64) (n / stride1);
    }

    return Upper ? (col - row >= k) : (col - row <= k);
  }

  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = mask(in) ? *in : 0;
  }

  __device__ __forceinline__ void operator()(float* v) {
    if (!mask(v))
      *v = 0;
  }

  const float *start;
  const int64 stride0, stride1, k;
};

void THCudaTensor_tril(THCState *state, THCudaTensor *self_, THCudaTensor *src_, int64 k)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCudaTensor *src = src_;
  if (self_ == src_)
    src = THCudaTensor_newContiguous(state, src_);

  int64 stride0 = src->stride[0];
  int64 stride1 = src->stride[1];
  float *start = THCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<0> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCudaTensor_freeCopyTo(state, src, src_);

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_triu(THCState *state, THCudaTensor *self_, THCudaTensor *src_, int64 k)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src_));
  THArgCheck(src_->nDimension == 2, 1, "expected a matrix");

  THCudaTensor *src = src_;
  if (self_ == src_)
    src = THCudaTensor_newContiguous(state, src_);

  int64 stride0 = src->stride[0];
  int64 stride1 = src->stride[1];
  float *start = THCudaTensor_data(state, src) + src->storageOffset;

  TensorTriOp<1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THCudaTensor_pointwiseApply1(state, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THCudaTensor_pointwiseApply2(state, self_, src, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  if (self_ == src_)
    THCudaTensor_freeCopyTo(state, src, src_);

  THCudaCheck(hipGetLastError());
}
